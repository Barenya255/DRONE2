#include "hip/hip_runtime.h"
#include "stdio.h"
extern "C" {
#include "graph.h"
#include "algorithm/common.h"
}

#define CHECK(call)                                   \
do                                                    \
{                                                     \
    const hipError_t error_code = call;              \
    if (error_code != hipSuccess)                    \
    {                                                 \
        printf("CUDA Error:\n");                      \
        printf("    File:       %s\n", __FILE__);     \
        printf("    Line:       %d\n", __LINE__);     \
        printf("    Error code: %d\n", error_code);   \
        printf("    Error text: %s\n",                \
            hipGetErrorString(error_code));          \
        exit(1);                                      \
    }                                                 \
} while (0)

int addVertex(bool *exist, int *Global2Local, int *Local2Global, int localVertexSize, int u) {
    exist[u] = true;
    Global2Local[u] = localVertexSize;
    Local2Global[localVertexSize] = u;
    return localVertexSize + 1;
}

Graph* build_graph(int globalVertexSize, int edgeSize, int *u, int *v, int workerId, int workerNum, Comm *comm) {
    ncclUniqueId id;
    if (workerId == 0) {
        ncclGetUniqueId(&id);
        FILE *fp = fopen("./nccl.id", "w");
        fwrite(&id, sizeof(id), 1, fp);
        fclose(fp);
    }

    int GID = workerId % 4;
    Graph *g = (Graph *) malloc(sizeof(Graph));
    g -> GID = GID;
    hipSetDevice(g -> GID);

    bool *exist;
    int *Global2Local, *Local2Global;
    int localVertexSize = 0;
    hipHostMalloc((void **) &exist, sizeof(bool) * globalVertexSize);
    hipHostMalloc((void **) &Global2Local, sizeof(int) * globalVertexSize);
    hipHostMalloc((void **) &Local2Global, sizeof(int) * globalVertexSize);

    hipDeviceSynchronize();
    for (int i = 0; i < globalVertexSize; i++) {
        exist[i] = false;
        Global2Local[i] = -1;
    }

    for (int i = 0; i < edgeSize; i++) {
        if (!exist[u[i]]) localVertexSize = addVertex(exist, Global2Local, Local2Global, localVertexSize, u[i]);
        if (!exist[v[i]]) localVertexSize = addVertex(exist, Global2Local, Local2Global, localVertexSize, v[i]);
    }

    int *localOutDegree;
    hipHostMalloc((void **) &localOutDegree, sizeof(int) * localVertexSize);
    for (int i = 0; i < localVertexSize; i++) localOutDegree[i] = 0;
    for (int i = 0; i < edgeSize; i++) {
        localOutDegree[Global2Local[u[i]]]++;
    }

    int *addDiff;
    hipHostMalloc((void **) &addDiff, sizeof(int) * localVertexSize);
    for (int i = 0; i < localVertexSize; i++) addDiff[i] = 0;

    int *index, *dst;
    float *edgeLenHost;
    hipHostMalloc((void **) &index, sizeof(int) * (localVertexSize + 1));
    hipHostMalloc((void **) &dst, sizeof(int) * edgeSize);
    hipHostMalloc((void **) &edgeLenHost, sizeof(float) * edgeSize);
    index[0] = 0;
    for (int i = 1; i <= localVertexSize; i++) index[i] = index[i - 1] + localOutDegree[i - 1];
    for (int i = 0; i < edgeSize; i++) {
        int local_u = Global2Local[u[i]];
        int local_v = Global2Local[v[i]];
        dst[index[local_u] + addDiff[local_u]] = local_v;
        edgeLenHost[index[local_u] + addDiff[local_u]] = 1.0;
        addDiff[local_u]++;
    }

    hipMalloc((void **) &(g->Global2Local), sizeof(int) * globalVertexSize);
    hipMalloc((void **) &(g->Local2Global), sizeof(int) * localVertexSize);
    hipMalloc((void **) &(g->index), sizeof(int) * (localVertexSize + 1));
    hipMalloc((void **) &(g->dst), sizeof(int) * edgeSize);
    hipMalloc((void **) &(g->edgeLen), sizeof(float) * edgeSize);

    hipMallocManaged((void **) &(g->localVertexSize), sizeof(int));
    hipMallocManaged((void **) &(g->globalVertexSize), sizeof(int));
    hipMallocManaged((void **) &(g->edgeSize), sizeof(int));

    hipMemcpy(g->Global2Local, Global2Local, sizeof(int) * globalVertexSize, hipMemcpyHostToDevice);
    hipMemcpy(g->Local2Global, Local2Global, sizeof(int) * localVertexSize, hipMemcpyHostToDevice);
    hipMemcpy(g->index, index, sizeof(int) * (localVertexSize + 1), hipMemcpyHostToDevice);
    hipMemcpy(g->dst, dst, sizeof(int) * edgeSize, hipMemcpyHostToDevice);
    hipMemcpy(g->edgeLen, edgeLenHost, sizeof(float) * edgeSize, hipMemcpyHostToDevice);
    g->localVertexSize[0] = localVertexSize;
    g->globalVertexSize[0] = globalVertexSize;
    g->edgeSize[0] = edgeSize;

    hipHostFree(exist);
    hipHostFree(Global2Local);
    hipHostFree(Local2Global);
    hipHostFree(localOutDegree);
    hipHostFree(addDiff);
    hipHostFree(index);
    hipHostFree(dst);
    hipHostFree(edgeLenHost);

    g->blockSize = 256;
    g->gridSize = (*g->localVertexSize - 1) / g->blockSize + 1;
    if (g -> gridSize > 1024) {
        g -> gridSize = 1024;
    }

    if (workerId != 0) {
        FILE *fp = fopen("./nccl.id", "r");
        fread(&id, sizeof(id), 1, fp);
        fclose(fp);
    }

    NCCLCHECK(ncclCommInitRank(&comm->comm, workerNum, id, workerId));
    hipStreamCreate(&comm->s);

    return g;
}

Graph* build_graph_withLen(int globalVertexSize, int edgeSize, int *u, int *v, float *edgeLen) {
    const size_t malloc_limit = size_t(1024) * size_t(1024) * size_t(1024) * 5;
    hipDeviceSetLimit(hipLimitMallocHeapSize, malloc_limit);

    Graph *g = (Graph *) malloc(sizeof(Graph));
    bool *exist;
    int *Global2Local, *Local2Global;
    int localVertexSize = 0;
    hipHostMalloc((void **) &exist, sizeof(bool) * globalVertexSize);
    hipHostMalloc((void **) &Global2Local, sizeof(int) * globalVertexSize);
    hipHostMalloc((void **) &Local2Global, sizeof(int) * globalVertexSize);

    hipDeviceSynchronize();
    for (int i = 0; i < globalVertexSize; i++) {
        exist[i] = false;
        Global2Local[i] = -1;
    }

//    printf("cgo, edgeSize:%d\n", edgeSize);
//    printf("cgo, u[2994436]:%d, v[2994436]:%d\n", u[2994436], v[2994436]);

    for (int i = 0; i < edgeSize; i++) {
//        if (u[i] == 1386507 || v[i] == 1386507) {
//            printf("u[%d]:%d, v[%d]:%d\n", i, u[i], i, v[i]);
//        }
        if (!exist[u[i]]) localVertexSize = addVertex(exist, Global2Local, Local2Global, localVertexSize, u[i]);
        if (!exist[v[i]]) localVertexSize = addVertex(exist, Global2Local, Local2Global, localVertexSize, v[i]);
    }

//    printf("build: Global2Local[1386507]:%d\n", Global2Local[1386507]);

    int *localOutDegree;
    hipHostMalloc((void **) &localOutDegree, sizeof(int) * localVertexSize);
    for (int i = 0; i < localVertexSize; i++) localOutDegree[i] = 0;
    for (int i = 0; i < edgeSize; i++) {
        localOutDegree[Global2Local[u[i]]]++;
    }

    int *addDiff;
    hipHostMalloc((void **) &addDiff, sizeof(int) * localVertexSize);
    for (int i = 0; i < localVertexSize; i++) addDiff[i] = 0;

    int *index, *dst;
    float *edgeLenHost;
    hipHostMalloc((void **) &index, sizeof(int) * (localVertexSize + 1));
    hipHostMalloc((void **) &dst, sizeof(int) * edgeSize);
    hipHostMalloc((void **) &edgeLenHost, sizeof(float) * edgeSize);
    index[0] = 0;
    for (int i = 1; i <= localVertexSize; i++) index[i] = index[i - 1] + localOutDegree[i - 1];
    for (int i = 0; i < edgeSize; i++) {
        int local_u = Global2Local[u[i]];
        int local_v = Global2Local[v[i]];
        dst[index[local_u] + addDiff[local_u]] = local_v;
        edgeLenHost[index[local_u] + addDiff[local_u]] = edgeLen[i];
        addDiff[local_u]++;
    }

    hipMalloc((void **) &(g->Global2Local), sizeof(int) * globalVertexSize);
    hipMalloc((void **) &(g->Local2Global), sizeof(int) * localVertexSize);
    hipMalloc((void **) &(g->index), sizeof(int) * (localVertexSize + 1));
    hipMalloc((void **) &(g->dst), sizeof(int) * edgeSize);
    hipMalloc((void **) &(g->edgeLen), sizeof(float) * edgeSize);

    hipMallocManaged((void **) &(g->localVertexSize), sizeof(int));
    hipMallocManaged((void **) &(g->globalVertexSize), sizeof(int));
    hipMallocManaged((void **) &(g->edgeSize), sizeof(int));

    hipMemcpy(g->Global2Local, Global2Local, sizeof(int) * globalVertexSize, hipMemcpyHostToDevice);
    hipMemcpy(g->Local2Global, Local2Global, sizeof(int) * localVertexSize, hipMemcpyHostToDevice);
    hipMemcpy(g->index, index, sizeof(int) * (localVertexSize + 1), hipMemcpyHostToDevice);
    hipMemcpy(g->dst, dst, sizeof(int) * edgeSize, hipMemcpyHostToDevice);
    hipMemcpy(g->edgeLen, edgeLenHost, sizeof(float) * edgeSize, hipMemcpyHostToDevice);
    g->localVertexSize[0] = localVertexSize;
    g->globalVertexSize[0] = globalVertexSize;
    g->edgeSize[0] = edgeSize;

    hipHostFree(exist);
    hipHostFree(Global2Local);
    hipHostFree(Local2Global);
    hipHostFree(localOutDegree);
    hipHostFree(addDiff);
    hipHostFree(index);
    hipHostFree(dst);
    hipHostFree(edgeLenHost);

    g->blockSize = 256;
    g->gridSize = (*g->localVertexSize - 1) / g->blockSize + 1;
    if (g -> gridSize > 1024) {
        g -> gridSize = 1024;
    }
    return g;
}


__global__ void calLocalMirrorNumber(int *Global2Local, int *masterVertex, int *mirrorNumber, int *MasterWorkerIndex, int masterSize) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;

    while (id < masterSize) {
        int localVertexId = Global2Local[masterVertex[id]];
        MasterWorkerIndex[localVertexId + 1] = mirrorNumber[id];
        id += stride;
    }
}

__global__ void setMaster2Workers(int *Master2Workers, int *MasterWorkerIndex, int *masterVertex, int *mirrorNumberSum, int *mirrorWorkers, int *Global2Local, int masterSize) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    while (id < masterSize) {
        int localVertexId = Global2Local[masterVertex[id]];
//        if (id % 1000 == 0) {
//            printf("cuda -> id:%d, masterVertex[id]:%d, localVertexId:%d\n", id, masterVertex[id], localVertexId);
//        }
        for (int i = 0; MasterWorkerIndex[localVertexId] + i < MasterWorkerIndex[localVertexId + 1]; i++) {
//            if (id == masterSize - 1) {
//                printf("cuda -> i:\n", i);
//                printf("cuda -> mirrorNumberSum[id] + i:\n", mirrorNumberSum[id] + i);
//            }
            Master2Workers[MasterWorkerIndex[localVertexId] + i] = mirrorWorkers[mirrorNumberSum[id] + i];
        }
        if (MasterWorkerIndex[localVertexId + 1] - MasterWorkerIndex[localVertexId] != mirrorNumberSum[id + 1] - mirrorNumberSum[id]) {
            printf("Error, index error for id:%d\n", id);
        }
        id += stride;
    }
}

void addMasterRoute(Graph* g, int *masterVertex, int *mirrorNumber, int *mirrorWorkers, int masterSize, int mirrorWorkerSize) {
//    int device;
//    hipGetDevice(&device);
//    printf("addMasterRoute: device: %d\n", device);
    hipSetDevice(g -> GID);

    g -> MasterSize = masterSize;
    g -> MirrorWorkerSize = mirrorWorkerSize;

    int *MasterWorkerIndex;
    int localVertexSize = getLocalVertexSize(g);
    CHECK(hipHostMalloc((void **) &MasterWorkerIndex, sizeof(int) * (localVertexSize + 1)));
    CHECK(hipMalloc((void **) &g->MasterWorkerIndex, sizeof(int) * (localVertexSize + 1)));

    int *masterVertexCUDA, *mirrorNumberCUDA, *mirrorWorkersCUDA;
    CHECK(hipMalloc((void **) &masterVertexCUDA, sizeof(int) * masterSize));
    CHECK(hipMalloc((void **) &mirrorNumberCUDA, sizeof(int) * masterSize));
    CHECK(hipMalloc((void **) &mirrorWorkersCUDA, sizeof(int) * mirrorWorkerSize));
    hipMemcpy(masterVertexCUDA, masterVertex, sizeof(int) * masterSize, hipMemcpyHostToDevice);
    hipMemcpy(mirrorNumberCUDA, mirrorNumber, sizeof(int) * masterSize, hipMemcpyHostToDevice);
    hipMemcpy(mirrorWorkersCUDA, mirrorWorkers, sizeof(int) * mirrorWorkerSize, hipMemcpyHostToDevice);

    calLocalMirrorNumber<<<g->gridSize, g->blockSize>>>(g->Global2Local, masterVertexCUDA, mirrorNumberCUDA,
                                                        g->MasterWorkerIndex, masterSize);
    hipDeviceSynchronize();
    CHECK(hipMemcpy(MasterWorkerIndex, g->MasterWorkerIndex, sizeof(int) * (localVertexSize + 1), hipMemcpyDeviceToHost));
    int *mirrorNumberSum;
    CHECK(hipMallocManaged((void **) &mirrorNumberSum, sizeof(int) * (localVertexSize + 1)));
    for (int i = 1; i <= localVertexSize; i++) {
        MasterWorkerIndex[i] += MasterWorkerIndex[i - 1];
        mirrorNumberSum[i] = mirrorNumberSum[i - 1] + mirrorNumber[i - 1];
    }
    hipMemcpy(g->MasterWorkerIndex, MasterWorkerIndex, sizeof(int) * (localVertexSize + 1), hipMemcpyHostToDevice);

    CHECK(hipMalloc((void **) &g->Master2Workers, sizeof(int) * mirrorWorkerSize));
//    printf("masterSize:%d\n", masterSize);
    setMaster2Workers<<<g->gridSize, g->blockSize>>>(g->Master2Workers, g->MasterWorkerIndex, masterVertexCUDA,
                      mirrorNumberSum, mirrorWorkersCUDA, g->Global2Local, masterSize);
    hipDeviceSynchronize();

    CHECK(hipFree(masterVertexCUDA));
    CHECK(hipFree(mirrorNumberCUDA));
    hipFree(mirrorWorkersCUDA);
    hipHostFree(MasterWorkerIndex);
    CHECK(hipFree(mirrorNumberSum));
}

__global__ void setMirror2Worker(int *Mirror2Worker, int *mirrorVertex, int *masterWorker, int *Global2Local, int mirrorSize) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    while (id < mirrorSize) {
        int localVertexId = Global2Local[mirrorVertex[id]];
        Mirror2Worker[localVertexId] = masterWorker[id];
        id += stride;
    }
}

void addMirrorRoute(Graph* g, int *mirrorVertex, int *masterWorker, int mirrorSize) {
    hipSetDevice(g -> GID);
    g -> MirrorSize = mirrorSize;

    int localVertexSize = getLocalVertexSize(g);
    CHECK(hipMallocManaged((void **) &g->Mirror2Worker, sizeof(int) * localVertexSize));
    for (int i = 0; i < localVertexSize; i++) g->Mirror2Worker[i] = -1;

    int *mirrorVertexCUDA, *masterWorkerCUDA;
    CHECK(hipMalloc(&mirrorVertexCUDA, sizeof(int) * mirrorSize));
    CHECK(hipMalloc(&masterWorkerCUDA, sizeof(int) * mirrorSize));
    hipMemcpy(mirrorVertexCUDA, mirrorVertex, sizeof(int) * mirrorSize, hipMemcpyHostToDevice);
    hipMemcpy(masterWorkerCUDA, masterWorker, sizeof(int) * mirrorSize, hipMemcpyHostToDevice);

    setMirror2Worker<<<g->gridSize, g->blockSize>>>(g->Mirror2Worker, mirrorVertexCUDA, masterWorkerCUDA,
                                                    g->Global2Local, mirrorSize);
    hipDeviceSynchronize();

    hipFree(mirrorVertexCUDA);
    hipFree(masterWorkerCUDA);
}

int getLocalVertexSize(Graph* g) {
    hipSetDevice(g -> GID);
//    int localVertexSize;
//    hipMemcpy(&localVertexSize, g -> localVertexSize, sizeof(int), hipMemcpyDeviceToHost);
//    return localVertexSize;
    return *g->localVertexSize;
}

int getGlobalVertexSize(Graph* g) {
    hipSetDevice(g -> GID);
    return *g->globalVertexSize;
}