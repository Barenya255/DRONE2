
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <hipsparse.h>

void print_vec(float *a, int len, char *name) {
	float* tmp;
	printf("%s:", name);
	hipHostMalloc(&tmp, sizeof(float) * len, hipHostMallocDefault);
	hipMemcpy(tmp, a, sizeof(float) * len, hipMemcpyDeviceToHost);
	for (int i = 0; i < len; i++) {
		printf(" %f", tmp[i]);
	}
	printf("\n");
	hipHostFree(tmp);
}

void csr2csc(hipsparseHandle_t handle, int *csrRowPtr, int *csrColInd, float* csrValues, int rows, int cols, int nnz) {
	size_t bufferSize;
	int *cscColPtr, *cscRowInd;
	float *cscValues;

	hipMallocManaged(&cscColPtr, sizeof(int) * (cols + 1));
	hipMallocManaged(&cscRowInd, sizeof(int) * nnz);
	hipMallocManaged(&cscValues, sizeof(float) * nnz);

	hipsparseCsr2cscEx2_bufferSize(handle, rows, cols, nnz, csrValues, csrRowPtr, csrColInd, cscValues, cscColPtr, cscRowInd,
								  HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, &bufferSize);
	float *buffer;
	printf("csr2csc buffer size needed: %d\n", bufferSize);
	hipMalloc(&buffer, bufferSize * sizeof(float));
	hipsparseCsr2cscEx2(handle, rows, cols, nnz, csrValues, csrRowPtr, csrColInd, cscValues, cscColPtr, cscRowInd,
								  HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, buffer);
	printf("cscColPtr:");
	for (int i = 0; i <= cols; i++) {
		printf(" %d", cscColPtr[i]);
	}
	printf("\ncscRowInd:");
	for (int i = 0; i < nnz; i++) {
		printf(" %d", cscRowInd[i]);
	}
	printf("\ncscValues:");
	for (int i = 0; i < nnz; i++) {
		printf(" %f", cscValues[i]);
	}
	printf("\n");
}

int main()
{
	hipsparseHandle_t handle = NULL;
	hipsparseCreate(&handle);

	hipsparseSpMatDescr_t matA;
	int *csrRowOffsets;
	int csrRowOffsets_host[] = {0, 2, 4, 5};
	hipMalloc(&csrRowOffsets, 4 * sizeof(int));
	hipMemcpy(csrRowOffsets, csrRowOffsets_host, 4 * sizeof(int), hipMemcpyHostToDevice);

	int nnz = 5;
	int *csrColInd;
	int csrColInd_host[] = {0, 2, 1, 3, 2};
	hipMalloc(&csrColInd, nnz * sizeof(int));
	hipMemcpy(csrColInd, csrColInd_host, nnz * sizeof(int), hipMemcpyHostToDevice);

	float *csrValues;
	float csrValues_host[] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f};
	hipMalloc(&csrValues, nnz * sizeof(float));
	hipMemcpy(csrValues, csrValues_host, nnz * sizeof(float), hipMemcpyHostToDevice);


	hipsparseStatus_t status = hipsparseCreateCsr(&matA, 3, 4, nnz,
												csrRowOffsets, csrColInd, csrValues,
											  HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
											  HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
	csr2csc(handle, csrRowOffsets, csrColInd, csrValues, 3, 4, nnz);

	float *buffer;
	size_t bufferSize;





//	cusparseDnMatDescr_t matT;
//	float *tmp;
//	cudaMalloc(&tmp, 3 * 2 * sizeof(float));
//	cusparseCreateDnMat(&matT, 2, 3, 2, tmp,
//						CUDA_R_32F, CUSPARSE_ORDER_COL);
//	size_t bufferSize;
//	cusparseSparseToDense_bufferSize(handle, matA, matT,
//									 CUSPARSE_SPARSETODENSE_ALG_DEFAULT, &bufferSize);
//
//	printf("buffer size needed: %d\n", bufferSize);
//	float *buffer;
//	cudaMalloc(&buffer, bufferSize * sizeof(float));
//	cusparseSparseToDense(handle, matA, matT,
//									 CUSPARSE_SPARSETODENSE_ALG_DEFAULT, buffer);
//	print_vec(tmp, 6, "MatT:");

	hipsparseDnMatDescr_t matB, matC;

	float *values_host, *values;
	values_host = new float[16];
	for (int i = 0; i < 16; i++) values_host[i] = i;
	hipMalloc(&values, 16 * sizeof(float));
	hipMemcpy(values, values_host, 16 * sizeof(float), hipMemcpyHostToDevice);
	hipsparseCreateDnMat(&matB, 4, 4, 4, values, HIP_R_32F, HIPSPARSE_ORDER_COL);

	float *result, *result_host;
	hipMalloc(&result, 12 * sizeof(float));
	hipHostMalloc(&result_host, 12 * sizeof(float), hipHostMallocDefault);
//	cudaMemcpy(result, result_host, 4 * sizeof(float), cudaMemcpyHostToDevice);
	status = hipsparseCreateDnMat(&matC, 3, 4, 3, result, HIP_R_32F, HIPSPARSE_ORDER_COL);
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		printf("CUSPARSE API failed at line %d with error: %s (%d)\n",
			   __LINE__, hipsparseGetErrorString(status), status);
		exit(0);
	}

	float alpha = 1.0, beta = 0.0;
	status = hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, &beta, matC,
				 HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize);

//	if (status != CUSPARSE_STATUS_SUCCESS) {
//		printf("CUSPARSE API failed at line %d with error: %s (%d)\n",
//			   __LINE__, cusparseGetErrorString(status), status);
//		exit(0);
//	}
	printf("buffer size needed: %d\n", bufferSize);
	hipMalloc(&buffer, bufferSize * sizeof(float));

//	print_vec(result, 4, "result");
	status = hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, &beta, matC,
							HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, buffer);

//	if (status != CUSPARSE_STATUS_SUCCESS) {
//		printf("CUSPARSE API failed at line %d with error: %s (%d)\n",
//			   __LINE__, cusparseGetErrorString(status), status);
//		exit(0);
//	}
//	cusparseDestroySpMat(matA);
//	cusparseDestroyDnMat(matB);
//	cusparseDestroyDnMat(matC);
//	cusparseDestroy(handle);
//
//	print_vec(csrValues, nnz, "MatA:");
//	print_vec(values, 6, "MatB:");

//	print_vec(buffer, bufferSize, "buffer:");

	hipDeviceSynchronize();

//	printf("Result:");
//	cudaMemcpy(result_host, result, sizeof(float) * 14, cudaMemcpyDeviceToHost);
//	for (int i = 0; i < 4; i++) {
//		printf(" %f", result_host[i]);
//	}
//	printf("\n");

	print_vec(result, 12, "result");
//	cusparseDnMatGetValues(matC, (void**)&result);
//	print_vec(result, 4, "ans");
}