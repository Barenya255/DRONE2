#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>

// m * n (row major) -> n * m (column major) transpose
void Transpose(hipblasHandle_t handle, int m, int n, float *d_A, float *d_A_T)
{
    float alpha = 1.0f;
    float beta = 0.0f;

    hipblasSgeam(handle,
                HIPBLAS_OP_T, HIPBLAS_OP_N,
                m, n,
                &alpha,
                d_A, n,
                &beta,
                d_A, m,
                d_A_T, m);
}

void printPlainMatrix(const float* matrix, const int H, const int W)
{
    for (int i = 0; i < H; i++)
    {
        for (int j = 0; j < W; j++)
        {
            std::cout << std::fixed << std::setw(8) << std::setprecision(4) << matrix[i * W + j];
        }
        std::cout << std::endl;
    }
}

//void csr2bsr() {
//    // Suppose that A is m x n sparse matrix represented by CSR format,
//    // hx is a host vector of size n, and hy is also a host vector of size m.
//    // m and n are not multiple of blockDim.
//    // step 1: transform CSR to BSR with column-major order
//    int base, nnz;
//    int nnzb;
//    hipsparseDirection_t dirA = HIPSPARSE_DIRECTION_COLUMN;
//    int mb = (m + blockDim-1)/blockDim;
//    int nb = (n + blockDim-1)/blockDim;
//    hipMalloc((void**)&bsrRowPtrC, sizeof(int) *(mb+1));
//    hipsparseXcsr2bsrNnz(handle, dirA, m, n,
//                        descrA, csrRowPtrA, csrColIndA, blockDim,
//                        descrC, bsrRowPtrC, &nnzb);
//    hipMalloc((void**)&bsrColIndC, sizeof(int)*nnzb);
//    hipMalloc((void**)&bsrValC, sizeof(float)*(blockDim*blockDim)*nnzb);
//    hipsparseScsr2bsr(handle, dirA, m, n,
//                     descrA, csrValA, csrRowPtrA, csrColIndA, blockDim,
//                     descrC, bsrValC, bsrRowPtrC, bsrColIndC);
//}

// Example usage
int main()
{
    const int n = 2;
    const int m = 3;
    const int k = 4;

    float *b, *c, *bt;
    float alpha, beta;
    alpha = 1.0;
    beta = 0.0;

    // Create a Cublas handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipsparseHandle_t sparse_handle;
    hipsparseCreate(&sparse_handle);

    hipMallocManaged(&b, sizeof(float) * m * k);
    hipMallocManaged(&c, sizeof(float) * m * k);
    hipMallocManaged(&bt, sizeof(float) * m * k);

    for (int i = 0; i < m; i++) for (int j = 0; j < k; j++) {
        b[i * k + j] = i * k + j;
    }
    printf("B ori:\n");
    printPlainMatrix(b, m, k);

    Transpose(handle, m, k, b, bt);
    hipDeviceSynchronize();
    printf("B transpose:\n");
    printPlainMatrix(bt, k, m);

    int nnz = 3;
    float *csr_val, *csrRowPtr, *csrColInd;
    hipMallocManaged(&csr_val, sizeof(float) * nnz);
    csr_val[0] = 1; csr_val[1] = 2; csr_val[2] = 3;
    hipMallocManaged(&csrRowPtr, sizeof(float) * (n + 1));
    csrRowPtr[0] = 0;
    csrRowPtr[1] = 2;
    csrRowPtr[2] = 3;
    hipMallocManaged(&csrColInd, sizeof(float) * nnz);
    csrColInd[0] = 0; csrColInd[1] = 2; csrColInd[2] = 1;

    hipsparseMatDescr_t descr;
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

    cusparseScsrmm_batched(sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, k, m, nnz, &alpha, descr,csr_val,csrRowPtr,csrColInd, bt, m, &beta,c, n);

    printf("C:\n");
    printPlainMatrix(c, n, k);

    // Destroy the Cublas handle and free memory on the device
    hipblasDestroy(handle);
    hipsparseDestroy(sparse_handle);

    return 0;
}